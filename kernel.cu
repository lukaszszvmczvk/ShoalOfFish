#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>

#include <thrust/gather.h>

unsigned int block_size = 256;

constexpr float max_speed = 15.0f;

constexpr float turn_factor = 1.f;

constexpr float height = 900.f;
constexpr float width = 1600.f;

constexpr float margin = 50;


__global__ void update_pos(Fish* fishes, unsigned int N, float visualRange, float minDistance, float rule1_scale, float rule2_scale, float rule3_scale)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }


    int neighbour_count = 0;
    float centerX = 0.f;
    float centerY = 0.f;
    float moveX = 0.f;
    float moveY = 0.f;
    float avgDX = 0.f;
    float avgDY = 0.f;
    
    float dx = fishes[index].dxP;
    float dy = fishes[index].dyP;

    for (int i = 0; i < N; ++i)
    {
        if (i == index)
            continue;
        float distance = glm::sqrt((fishes[index].x - fishes[i].x) * (fishes[index].x - fishes[i].x) +
            (fishes[index].y - fishes[i].y) * (fishes[index].y - fishes[i].y));
        if (distance < visualRange)
        {
            ++neighbour_count;
            centerX += fishes[i].x;
            centerY += fishes[i].y;
            avgDX += fishes[i].dx;
            avgDY += fishes[i].dy;

            if (distance < minDistance)
            {
                moveX += fishes[index].x - fishes[i].x;
                moveY += fishes[index].y - fishes[i].y;
            }
        }
    }

    if (neighbour_count > 0)
    {
        // rule1 Coherence
        centerX = centerX / neighbour_count;
        centerY = centerY / neighbour_count;

        dx += (centerX - fishes[index].x) * rule1_scale;
        dy += (centerY - fishes[index].y) * rule1_scale;

        // rule2 
        dx += moveX * rule2_scale;
        dy += moveY * rule2_scale;

        // rule3
        avgDX = avgDX / neighbour_count;
        avgDY = avgDY / neighbour_count;

        dx += (avgDX - fishes[index].dxP) * rule3_scale;
        dy += (avgDY - fishes[index].dyP) * rule3_scale;
    }


    float speed = glm::sqrt(dx * dx + dy * dy);
    if (speed > max_speed)
    {
        dx = (dx / speed) * max_speed;
        dy = (dy / speed) * max_speed;
    }

    if (fishes[index].x < margin)
        dx += turn_factor;
    if (fishes[index].y < margin)
        dy += turn_factor;
    if (fishes[index].x > width - margin)
        dx -= turn_factor;
    if (fishes[index].y > height - margin)
        dy -= turn_factor;

    fishes[index].dx = dx;
    fishes[index].dy = dy;

    fishes[index].x += fishes[index].dx;
    fishes[index].y += fishes[index].dy;
}

void Boids::init_simulation(unsigned int N)
{
    Fish* fishes = nullptr;

    dim3 fullBlocksPerGrid((N + block_size - 1) / block_size);

    hipMalloc(reinterpret_cast<void**>(&fishes), N * sizeof(Fish));

    hipDeviceSynchronize();
}

void Boids::update_fishes(Fish* fishes, unsigned int N, float vr, float md, float r1, float r2, float r3)
{
    const dim3 full_blocks_per_grid((N + block_size - 1) /
        block_size);
    const dim3 threads_per_block(block_size);

    Fish* fishes_gpu = 0;
    hipSetDevice(0);
    hipMalloc(reinterpret_cast<void**>(&fishes_gpu), N * sizeof(Fish));
    hipMemcpy(fishes_gpu, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    update_pos << <full_blocks_per_grid, threads_per_block >> > (fishes_gpu, N, vr, md, r1, r2, r3);

    hipDeviceSynchronize();
    hipMemcpy(fishes, fishes_gpu, N * sizeof(Fish), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; ++i)
    {
        fishes[i].dxP = fishes[i].dx;
        fishes[i].dyP = fishes[i].dy;
    }
}