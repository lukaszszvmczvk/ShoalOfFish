#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include "kernel.h"
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>

#include <thrust/gather.h>

unsigned int block_size = 256;
constexpr float max_speed = 20.0f;
constexpr float turn_factor = 1.f;
constexpr float height = 900.f;
constexpr float width = 1600.f;
constexpr float margin = 50;

glm::vec2* positions = nullptr;
glm::vec2* velocity1 = nullptr;
glm::vec2* velocity2 = nullptr;
Fish* fishes_gpu = nullptr;
Fish* fishes_gpu_sorted = nullptr;
int* indices = nullptr;
int* grid_cell_indices = nullptr;
glm::vec2* pos_sorted = nullptr;
glm::vec2* vel_sorted = nullptr;


__global__ void update_vel(glm::vec2* pos, glm::vec2* vel1, glm::vec2* vel2, Fish* fishes, int* grid_cell_indices, int* grid_cell_start, int* grid_cell_end,
    unsigned int N, float visualRange, float minDistance, float cohesion_scale, float separation_scale, float alignment_scale)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    int neighbour_count = 0;
    float centerX = 0.f;
    float centerY = 0.f;
    float moveX = 0.f;
    float moveY = 0.f;
    float avgDX = 0.f;
    float avgDY = 0.f;
    
    float dx = vel1[index].x;
    float dy = vel1[index].y;

    int cell_index = grid_cell_indices[index];

    int row_cells = width / (2 * visualRange) + 1;
    int grid_size = row_cells * (height / (2 * visualRange) + 1);

    int neighbour_cells[] = { index + 1, index - 1, index - row_cells, index + row_cells, index + row_cells - 1, index + row_cells + 1,
                index - row_cells + 1, index - row_cells - 1 };

    for (int j = 0; j < 8; ++j)
    {
        int current_cell = neighbour_cells[j];

        if (current_cell < 0 || current_cell >= grid_size)
            continue;

        for (int i = grid_cell_start[current_cell]; i < grid_cell_end[current_cell]; ++i)
        {
            if (i == index)
                continue;
            float distance = glm::sqrt((pos[index].x - pos[i].x) * (pos[index].x - pos[i].x) +
                (pos[index].y - pos[i].y) * (pos[index].y - pos[i].y));
            if (distance < visualRange)
            {
                ++neighbour_count;
                centerX += pos[i].x;
                centerY += pos[i].y;
                avgDX += vel1[i].x;
                avgDY += vel1[i].y;

                if (distance < minDistance)
                {
                    moveX += pos[index].x - pos[i].x;
                    moveY += pos[index].y - pos[i].y;
                }
            }
        }
    }

    if (neighbour_count > 0)
    {
        // rule1 Cohesion
        centerX = centerX / neighbour_count;
        centerY = centerY / neighbour_count;

        dx += (centerX - pos[index].x) * cohesion_scale;
        dy += (centerY - pos[index].y) * cohesion_scale;

        // rule2 Separation
        dx += moveX * separation_scale;
        dy += moveY * separation_scale;

        // rule3 Alignment
        avgDX = avgDX / neighbour_count;
        avgDY = avgDY / neighbour_count;

        dx += (avgDX - vel1[index].x) * alignment_scale;
        dy += (avgDY - vel1[index].y) * alignment_scale;
    }

    // keep fishes in bounds
    if (pos[index].x < margin)
        dx += turn_factor;
    if (pos[index].y < margin)
        dy += turn_factor;
    if (pos[index].x > width - margin)
        dx -= turn_factor;
    if (pos[index].y > height - margin)
        dy -= turn_factor;

    // check if speed is < max_speed
    float speed = glm::sqrt(dx * dx + dy * dy);
    if (speed > max_speed)
    {
        dx = (dx / speed) * max_speed;
        dy = (dy / speed) * max_speed;
    }

    // update velocities
    vel2[index].x = dx;
    vel2[index].y = dy;
}

__global__ void update_pos(glm::vec2* pos, glm::vec2* vel, unsigned int N)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    pos[index].x += vel[index].x;
    pos[index].y += vel[index].y;
}

__global__ void assign_grid_cell(glm::vec2* pos, int* grid_cells, int* indices, float cell_width, unsigned int N)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) { return; }

    float x = pos[index].x;
    float y = pos[index].y;

    int x_size = width / cell_width + 1;

    int x_cell = x / cell_width;
    int y_cell = y / cell_width;

    grid_cells[index] = y_cell * x_size + x_cell;
    indices[index] = index;
}

__global__ void compute_start_end_cell(int* grid_cell_indices, int* grid_cell_start, int* grid_cell_end, int grid_size, unsigned int N)
{
    const auto index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= grid_size) { return; }

    int start = -1;
    int i = 0;
    while (i < N && grid_cell_indices[i] <= index)
    {
        if (start == -1 && grid_cell_indices[i] == index)
        {
            start = i;
        }
        ++i;
    }
    if (start == -1)
    {
        grid_cell_start[index] = -1;
        grid_cell_end[index] = -1;
    }
    else
    {
        grid_cell_start[index] = start;
        grid_cell_end[index] = i;
    }
}

void Boids::init_simulation(unsigned int N)
{
    hipError_t cudaStatus;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
    }

    cudaStatus = hipMalloc(reinterpret_cast<void**>(&positions), N * sizeof(glm::vec2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&velocity1), N * sizeof(glm::vec2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&velocity2), N * sizeof(glm::vec2));
    if(cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&pos_sorted), N * sizeof(glm::vec2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&vel_sorted), N * sizeof(glm::vec2));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_indices), N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&indices), N * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&fishes_gpu), N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&fishes_gpu_sorted), N * sizeof(Fish));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    hipDeviceSynchronize();
}
void Boids::end_simulation()
{
    hipFree(positions);
    hipFree(velocity1);
    hipFree(velocity2);
    hipFree(indices);
    hipFree(grid_cell_indices);
    hipFree(fishes_gpu);
    hipFree(fishes_gpu_sorted);
}
void Boids::update_fishes(glm::vec2* pos, glm::vec2* vel, Fish* fishes, unsigned int N, float vr, float md, float r1, float r2, float r3)
{
    hipError_t cudaStatus;
    
    const dim3 full_blocks_per_grid((N + block_size - 1) / block_size);
    const dim3 threads_per_block(block_size);

    float cell_width = 2 * vr;
    int grid_size = (width / cell_width + 1) * (height / cell_width + 1);
    const dim3 full_blocks_per_grid2((grid_size + block_size - 1) / block_size);

    // Allocate memory for start and end indices
    int* grid_cell_start;
    int* grid_cell_end;

    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_start), grid_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }
    cudaStatus = hipMalloc(reinterpret_cast<void**>(&grid_cell_end), grid_size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
    }

    // Copy data to gpu
    cudaStatus = hipMemcpy(positions, pos, N * sizeof(glm::vec2), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(velocity1, vel, N * sizeof(glm::vec2), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(fishes_gpu, fishes, N * sizeof(Fish), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // Asign grid cell to every fish
    assign_grid_cell << <full_blocks_per_grid, threads_per_block >> > (positions, grid_cell_indices, indices, cell_width, N);
    hipDeviceSynchronize();

    // Cast arrays to perform thrust operations
    auto thrust_gci = thrust::device_pointer_cast(grid_cell_indices);
    auto thrust_i = thrust::device_pointer_cast(indices);
    auto thrust_p = thrust::device_pointer_cast(positions);
    auto thrust_v = thrust::device_pointer_cast(velocity1);
    auto thrust_ps = thrust::device_pointer_cast(pos_sorted);
    auto thrust_vs = thrust::device_pointer_cast(vel_sorted);
    auto thrust_f = thrust::device_pointer_cast(fishes_gpu);
    auto thrust_fs = thrust::device_pointer_cast(fishes_gpu_sorted);

    // Sort fishes indicies by grid cell
    thrust::sort_by_key(thrust_gci, thrust_gci + N, thrust_i);

    // Compute start and end indices of grid cell
    compute_start_end_cell << <full_blocks_per_grid2, threads_per_block >> > (grid_cell_indices, grid_cell_start, grid_cell_end, grid_size, N);
    hipDeviceSynchronize();

    // Sort fish pos and vel by indices
    thrust::gather(thrust_i, thrust_i + N, thrust_p, thrust_ps);
    thrust::gather(thrust_i, thrust_i + N, thrust_v, thrust_vs);
    thrust::gather(thrust_i, thrust_i + N, thrust_f, thrust_fs);


    update_vel << <full_blocks_per_grid, threads_per_block >> > (pos_sorted, vel_sorted, velocity2, fishes_gpu_sorted, grid_cell_indices, grid_cell_start, grid_cell_end,
        N, vr, md, r1, r2, r3);
    hipDeviceSynchronize();

    update_pos << <full_blocks_per_grid, threads_per_block >> > (pos_sorted, velocity2, N);
    hipDeviceSynchronize();
    

    // Przerzucenie pamieci na cpu
    cudaStatus = hipMemcpy(pos, pos_sorted, N * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(vel, velocity2, N * sizeof(glm::vec2), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }
    cudaStatus = hipMemcpy(fishes, fishes_gpu_sorted, N * sizeof(Fish), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
    }

    // Zwolnienie pami�ci
    hipFree(grid_cell_start);
    hipFree(grid_cell_end);
}